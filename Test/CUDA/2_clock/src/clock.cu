#include "hip/hip_runtime.h"
/*
 * clock.cu
 *
 *  Created on: Sep 3, 2014
 *      Author: chunk
 */

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define NUM_BLOCKS 64
#define NUM_THREADS 256
#define mmin(a,b) ((a)<(b) ? (a) : (b))

__global__ void timeReduction(int *A, clock_t* timer) {
	extern __shared__ int shared[];
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	if (tid == 0)
		timer[bid] = clock();
	shared[tid] = A[tid];
	shared[tid + blockDim.x] = A[tid + blockDim.x];

	for (unsigned int d = blockDim.x * 2; d > 0; d >> 1) {
		__syncthreads();
		if (tid < d)
			shared[tid] = mmin(shared[tid],shared[tid + d]);
	}
	__syncthreads();
	if (tid == 0)
		timer[bid + blockDim.x] = clock();
}

int main(int argc, char **argv) {
	int devID = findCudaDevice(argc, (const char**) argv);
	assert(devID >= 0);

	int n = NUM_THREADS * 2;
	int nbyte = n * sizeof(int);

	int *h_A, *d_A;
	clock_t *h_timer, *d_timer;

	hipHostMalloc(&h_A, nbyte);
	hipHostMalloc(&h_timer, NUM_BLOCKS * 2 * sizeof(clock_t));
	hipMalloc(&d_A, nbyte);
	hipMalloc(&d_timer, NUM_BLOCKS * 2 * sizeof(clock_t));
	srand(time(0));
	for (int i = 0; i < n; i++) {
		h_A[i] = (int) (NUM_THREADS * rand() / (RAND_MAX + 1.0));
	}
	hipMemset((void *) d_timer, 0, NUM_BLOCKS * 2 * sizeof(clock_t));
	hipMemcpy(d_A, h_A, nbyte, hipMemcpyHostToDevice);
	timeReduction<<<NUM_BLOCKS, NUM_THREADS, nbyte>>>(d_A, d_timer);
}

