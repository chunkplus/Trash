#include "hip/hip_runtime.h"
/*
 * asyncAPI.cu
 *
 *  Created on: Sep 3, 2014
 *      Author: chunk
 */

#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include "util.h"

__global__ void vecAdd(int* A, int* B, int* C) {
	int i = threadIdx.x;
	C[i] = A[i] + B[i];
}

void display(int *a, int length) {
	printf("\n");
	for (int i = 0; i < length; i++) {
		printf("%d ", a[i]);
	}
	printf("\n");
}

bool verify(int *a, int *b, int len) {
	for (int i = 0; i < len; i++) {
		if (a[i] != b[i])
			return false;
	}
	return true;
}

int main(int argc, char **argv) {
	int devId;
	hipDeviceProp_t devProp;
	devId = findCudaDevice(argc, (const char **) argv);
	hipGetDeviceProperties(&devProp, devId);

	printf("cuda device info : %d - [%s]\n", devId, devProp.name);

	const int n = 1024;
	const int nbyte = n * sizeof(int);
	int *h_A, *h_B, *h_C, *sum;
	int *d_A, *d_B, *d_C;

	hipHostMalloc(&h_A, nbyte);
	hipHostMalloc(&h_B, nbyte);
	hipHostMalloc(&h_C, nbyte);
//	h_C = (int *) malloc(nbyte);
	hipHostMalloc(&sum, nbyte);
	hipMalloc(&d_A, nbyte);
	hipMalloc(&d_B, nbyte);
	hipMalloc(&d_C, nbyte);
	hipMemset(&d_A, 0, nbyte);
	hipMemset(&d_B, 0, nbyte);
	hipMemset(&d_C, 0, nbyte);

	hipStream_t stream0, stream1, stream2;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	StopWatchInterface *timer = NULL;
	mTimer mtimer;
	cuTimer cutmer;
	sdkCreateTimer(&timer);

//	srand(time(0));
	for (int i = 0; i < n; i++) {
		h_A[i] = (int) (1024.0 * rand() / (RAND_MAX + 1.0));
		h_B[i] = (int) (1024.0 * rand() / (RAND_MAX + 1.0));
		sum[i] = h_A[i] + h_B[i];
	}

	sdkStartTimer(&timer);
	mtimer.start();
	cutmer.start();
	/**
	 *
	 * ____
	 * ____|___ ___ ___
	 */
	hipEventRecord(start, stream0);
	hipMemcpyAsync(d_A, h_A, nbyte, hipMemcpyHostToDevice, stream0);
	hipMemcpyAsync(d_B, h_B, nbyte, hipMemcpyHostToDevice, stream1);
//	hipStreamSynchronize(stream0);
//	hipStreamSynchronize(stream1);
//	vecAdd<<<1, n, 0, stream0>>>(d_A, d_B, d_C);
//	hipStreamSynchronize(stream0);
	display(h_C, n);
	checkCudaErrors(
			hipMemcpyAsync(h_C, d_C, nbyte, hipMemcpyDeviceToHost, stream0));
	hipStreamSynchronize(stream0);
	hipEventRecord(stop, stream0);
	display(h_C, n);
	cutmer.end();
	mtimer.end();
	sdkStopTimer(&timer);

	printf("time spent by CPU in CUDA calls: %.2f\n", sdkGetTimerValue(&timer));
	mtimer.showTime();
	cutmer.showTime();
	if (verify(h_C, sum, n))
		printf("Checking OK.\n");
	else
		printf("Checking Eroor!\n");

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipHostFree(h_A);
	hipHostFree(h_B);
	hipHostFree(h_C);
//	free(h_C);
	hipHostFree(sum);
	hipFree(d_A);
	hipFree(d_B);
	checkCudaErrors(hipFree((void * )d_C));
	checkCudaErrors(hipDeviceReset());
}

