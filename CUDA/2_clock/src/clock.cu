#include "hip/hip_runtime.h"
/*
 * clock.cu
 *
 *  Created on: Sep 3, 2014
 *      Author: chunk
 */

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define NUM_BLOCKS 64
#define NUM_THREADS 256
#define mmin(a,b) ((a)<(b) ? (a) : (b))

__global__ void timeReduction(int *A, clock_t* timer) {
	extern __shared__ int shared[];
	int tid = threadIdx.x;
	int bid = blockIdx.x;

//	printf("\n%d\n", blockDim.x);
	if (tid == 0) {
		timer[bid] = clock();
//		printf("\n%d\n", timer[bid]);
	}
	shared[tid] = A[tid];
	shared[tid + blockDim.x] = A[tid + blockDim.x];

	for (unsigned int d = blockDim.x; d > 0; d /= 2) {
		__syncthreads();
		if (tid < d)
			shared[tid] = mmin(shared[tid],shared[tid + d]);
	}
	__syncthreads();
	A[tid] = shared[tid];

	if (tid == 0) {
//		timer[bid + blockDim.x] = clock();	// HERE : bug eye!
		timer[bid + gridDim.x] = clock();
//		printf("\n%d\n", shared[tid]);
//		printf("\n\t%d\n", timer[bid + blockDim.x]);
	}
}

void display(int *a, int length) {
	printf("\n");
	for (int i = 0; i < length; i++) {
		printf("%d ", a[i]);
	}
	printf("\n");
}
void display(clock_t *a, int length) {
	printf("\n");
	for (int i = 0; i < length; i++) {
		printf("%d ", a[i]);
	}
	printf("\n");
}

int main(int argc, char **argv) {
	int devID = findCudaDevice(argc, (const char**) argv);
	assert(devID >= 0);

	int n = NUM_THREADS * 2;
	int nbyte = n * sizeof(int);

	int *h_A, *d_A;
	clock_t *h_timer, *d_timer;

	hipHostMalloc(&h_A, nbyte);
	hipHostMalloc(&h_timer, NUM_BLOCKS * 2 * sizeof(clock_t));
	hipMalloc(&d_A, nbyte);
	hipMalloc(&d_timer, NUM_BLOCKS * 2 * sizeof(clock_t));
	srand(time(0));
	for (int i = 0; i < n; i++) {
		h_A[i] = (int) ((float) NUM_THREADS * rand() / (RAND_MAX + 1.0));
	}
//	display(h_A, n);

	checkCudaErrors(
			hipMemset((void *) d_timer, 0, NUM_BLOCKS * 2 * sizeof(clock_t)));
	checkCudaErrors(hipMemcpy(d_A, h_A, nbyte, hipMemcpyHostToDevice));
	timeReduction<<<NUM_BLOCKS, NUM_THREADS, nbyte>>>(d_A, d_timer);
	checkCudaErrors(hipPeekAtLastError());
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipMemcpy(h_A, d_A, nbyte, hipMemcpyDeviceToHost));
	checkCudaErrors(
			hipMemcpy((void *)h_timer, (const void *)d_timer, sizeof(clock_t) * NUM_BLOCKS * 2, hipMemcpyDeviceToHost));

	display(h_A, n);
	display(h_timer, NUM_BLOCKS * 2);

	clock_t tmin = h_timer[0], tmax = h_timer[NUM_BLOCKS];
	for (int i = 0; i < NUM_BLOCKS; i++) {
		if (h_timer[i] < tmin)
			tmin = h_timer[i];
		if (h_timer[i + NUM_BLOCKS] > tmax)
			tmax = h_timer[i + NUM_BLOCKS];
	}
	printf("\n%d\n", (int) (tmax - tmin));

	checkCudaErrors(hipHostFree(h_A));
	checkCudaErrors(hipHostFree(h_timer));
	checkCudaErrors(hipFree(d_A));
	checkCudaErrors(hipFree(d_timer));
	hipDeviceReset();
}

