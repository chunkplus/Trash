#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cutil_inline.h"

#include <stdio.h>
#include <math.h>

static void HandleError( hipError_t err,const char *file,int line ) 
{

  if (err != hipSuccess)

  {

    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
      file, line );
    exit( EXIT_FAILURE );
  }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
#define N (1024*1024)
#define FULL_DATA_SIZE N*20

__global__ void kernel(int* a, int *b, int*c)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = gridDim.x * blockDim.x;

  if (idx < N)
  {
    int idx1 = (idx + 1) % 256;
    int idx2 = (idx + 2) % 256;
    float as = (a[idx] + a[idx1] + a[idx2]) / 3;
    float bs = (b[idx] + b[idx1] + b[idx2]) / 3;
    c[idx] = (as + bs) / 2;
  }
}

int main()
{
  hipDeviceProp_t prop;
  int devID;

  HANDLE_ERROR(hipGetDevice(&devID));
  HANDLE_ERROR(hipGetDeviceProperties(&prop, devID));

  if (!prop.deviceOverlap)
  {
    printf("No device will handle overlaps. so no speed up from stream.\n");
    return 0;
  }

  hipEvent_t start, stop;
  float elapsedTime;
  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, 0));

  hipStream_t stream0;
  hipStream_t stream1;
  HANDLE_ERROR(hipStreamCreate(&stream0));
  HANDLE_ERROR(hipStreamCreate(&stream1));

  int *host_a, *host_b, *host_c;
  int *dev_a0, *dev_b0, *dev_c0;
  int *dev_a1, *dev_b1, *dev_c1;

  HANDLE_ERROR(hipMalloc((void**)&dev_a0, N*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_b0, N*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_c0, N*sizeof(int)));

  HANDLE_ERROR(hipMalloc((void**)&dev_a1, N*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_b1, N*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_c1, N*sizeof(int)));

  HANDLE_ERROR(hipHostAlloc((void**)&host_a, FULL_DATA_SIZE*sizeof(int), hipHostMallocDefault));
  HANDLE_ERROR(hipHostAlloc((void**)&host_b, FULL_DATA_SIZE*sizeof(int), hipHostMallocDefault));
  HANDLE_ERROR(hipHostAlloc((void**)&host_c, FULL_DATA_SIZE*sizeof(int), hipHostMallocDefault));


  for (int i=0; i<FULL_DATA_SIZE; i++)
  {
    host_a[i] = rand();
    host_b[i] = rand();
  }


  // tasks are put into stack for gpu execution
  for (int i=0; i<FULL_DATA_SIZE; i+=2*N)
  {
    HANDLE_ERROR(hipMemcpyAsync(dev_a0, host_a+i, N*sizeof(int), hipMemcpyHostToDevice, stream0));
    HANDLE_ERROR(hipMemcpyAsync(dev_a1, host_a+i+N, N*sizeof(int), hipMemcpyHostToDevice, stream1));

    HANDLE_ERROR(hipMemcpyAsync(dev_b0, host_b+i, N*sizeof(int), hipMemcpyHostToDevice, stream0));
    HANDLE_ERROR(hipMemcpyAsync(dev_b1, host_a+i+N, N*sizeof(int), hipMemcpyHostToDevice, stream1));

    kernel<<<N/256, 256, 0, stream0>>>(dev_a0, dev_b0, dev_c0);
    kernel<<<N/256, 256, 0, stream1>>>(dev_a1, dev_b1, dev_c1);

    HANDLE_ERROR(hipMemcpyAsync(host_c+i, dev_c0, N*sizeof(int), hipMemcpyDeviceToHost, stream0));
    HANDLE_ERROR(hipMemcpyAsync(host_c+i+N, dev_c1, N*sizeof(int), hipMemcpyDeviceToHost, stream1));

// 		HANDLE_ERROR(hipMemcpyAsync(dev_a0, host_a+i, N*sizeof(int), hipMemcpyHostToDevice, stream0));
// 		HANDLE_ERROR(hipMemcpyAsync(dev_b0, host_b+i, N*sizeof(int), hipMemcpyHostToDevice, stream0));
// 		kernel<<<N/256, 256, 0, stream0>>>(dev_a0, dev_b0, dev_c0);
// 		HANDLE_ERROR(hipMemcpyAsync(host_c+i, dev_c0, N*sizeof(int), hipMemcpyDeviceToHost, stream0));
// 
// 		HANDLE_ERROR(hipMemcpyAsync(dev_a1, host_a+i+N, N*sizeof(int), hipMemcpyHostToDevice, stream1));
// 		HANDLE_ERROR(hipMemcpyAsync(dev_a1, host_a+i+N, N*sizeof(int), hipMemcpyHostToDevice, stream1));
// 		kernel<<<N/256, 256, 0, stream1>>>(dev_a1, dev_b1, dev_c1);
// 		HANDLE_ERROR(hipMemcpyAsync(host_c+i+N, dev_c1, N*sizeof(int), hipMemcpyDeviceToHost, stream1));
  }

  // wait until gpu execution finish
  HANDLE_ERROR(hipStreamSynchronize(stream0));
  HANDLE_ERROR(hipStreamSynchronize(stream1));

  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

  printf("Time taken: %3.1f ms\n", elapsedTime);

  // free stream and mem
  HANDLE_ERROR(hipHostFree(host_a));
  HANDLE_ERROR(hipHostFree(host_b));
  HANDLE_ERROR(hipHostFree(host_c));
  HANDLE_ERROR(hipFree(dev_a0));
  HANDLE_ERROR(hipFree(dev_b0));
  HANDLE_ERROR(hipFree(dev_c0));
  HANDLE_ERROR(hipFree(dev_a1));
  HANDLE_ERROR(hipFree(dev_b1));
  HANDLE_ERROR(hipFree(dev_c1));
  HANDLE_ERROR(hipStreamDestroy(stream0));
  HANDLE_ERROR(hipStreamDestroy(stream1));
  return 0;
}
